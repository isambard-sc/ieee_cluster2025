/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 *    list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 *    contributors may be used to endorse or promote products derived from
 *    this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// function to add the elements of two arrays

__global__ void add(int n, float *x, float *y)
{
  int i = threadIdx.x;
  if (i < n) y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 128;

  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++)
  {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // optional hint
  // cudaMemPrefetchAsync(y, N * sizeof(float), 0);

  add<<<1, 128 >>>(N, x, y);
  hipDeviceSynchronize();

  for (int i = 0; i < N; i++)
  {
    printf("%f", y[i]);
  }

  hipFree(x);
  hipFree(y);

  return 0;
}
